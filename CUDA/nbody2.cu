#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>
#define SOFTENING 1e-9f
#define BLOCK_SIZE 128
#define BLOCK_STEP 32
#define BLOCK_NUM 32
#define MOD(a,b) ((a) - (a) / (b) * (b))
/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */
typedef struct { float x, y, z, vx, vy, vz; } Body;
/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float* data, int n) {
    for (int i = 0; i < n; i++) {
        data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
    }
}
/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */
__global__ void bodyForce(Body* p, float dt, int n) {
    //ÿ���߳���һ�����һ����
    int i = MOD(threadIdx.x + blockIdx.x * blockDim.x, n);//�̸߳�����������
    int nn = n / (BLOCK_STEP * BLOCK_SIZE);//�����߳������Ӻ���߳������ѭ������
    float Fx = 0.0f;
    float Fy = 0.0f;
    float Fz = 0.0f;
    __shared__ float3 p_shared[BLOCK_SIZE];//�����߳̿����߳̿��С�Ĺ����ڴ�
    float xi = p[i].x;
    float yi = p[i].y;
    float zi = p[i].z;//�����Ӧλ�õ�����
    float dx, dy, dz, distSqr, invDist, invDist3;
    int loc;
 //ѭ��չ��
#pragma unroll 32
    for (register int j = (blockIdx.x + blockIdx.x / BLOCK_NUM); nn-- > 0; j += BLOCK_STEP) {
        //�ӵ�ǰ�鿪ʼ��ǰ�ƽ�
        j = MOD(j, BLOCK_NUM);
        //��õ�ǰ���еĵ�ǰ���λ�õ����ݣ���д��shared��
        loc = j * BLOCK_SIZE + threadIdx.x;
        p_shared[threadIdx.x] = make_float3(p[loc].x, p[loc].y, p[loc].z);
        //ͬ������ֹ�����ڴ�δ�޸���ɾ�ʹ��
        __syncthreads();
 //ѭ��չ��
#pragma unroll 32
        //�̼߳����为��������������߳̿��С�����������������
        for (register int k = 0; k < BLOCK_SIZE; k++) {
            dx = p_shared[k].x - xi;
            dy = p_shared[k].y - yi;
            dz = p_shared[k].z - zi;
            distSqr = dx * dx + dy * dy + dz * dz + SOFTENING;
            invDist = rsqrtf(distSqr);
            invDist3 = invDist * invDist * invDist;
            Fx += dx * invDist3;
            Fy += dy * invDist3;
            Fz += dz * invDist3;
        }
        //ͬ������֤�����ڴ�ʹ����Ϻ���ܸ���
        __syncthreads();
    }
    //ԭ�Ӽӷ�ֹ���ݾ������³���
    atomicAdd(&p[i].vx, dt * Fx);
    atomicAdd(&p[i].vy, dt * Fy);
    atomicAdd(&p[i].vz, dt * Fz);
    if (blockIdx.x / BLOCK_NUM == BLOCK_STEP - 1) {
        atomicAdd(&p[i].x, p[i].vx * dt);
        atomicAdd(&p[i].y, p[i].vy * dt);
        atomicAdd(&p[i].z, p[i].vz * dt);
    }
}
int main(const int argc, const char** argv) {
    /*
     * Do not change the value for `nBodies` here. If you would like to modify it,
     * pass values into the command line.
     */
    int nBodies = 2 << 11;
    int salt = 1;
    if (argc > 1) nBodies = 2 << atoi(argv[1]);
    /*
     * This salt is for assessment reasons. Tampering with it will result in automatic failure.
     */
    if (argc > 2) salt = atoi(argv[2]);
    const float dt = 0.01f; // time step
    const int nIters = 10;  // simulation iterations
    int bytes = nBodies * sizeof(Body);
    float* buf;
    hipHostMalloc((void**)&buf, bytes);
    /*
     * As a constraint of this exercise, `randomizeBodies` must remain a host function.
     */
    randomizeBodies(buf, 6 * nBodies); // Init pos / vel data
    float* d_buf;
    hipMalloc((void**)&d_buf, bytes);
    Body* d_p = (Body*)d_buf;
    hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
    double totalTime = 0.0;
    /*
     * This simulation will run for 10 cycles of time, calculating gravitational
     * interaction amongst bodies, and adjusting their positions to reflect.
     */
     /*******************************************************************/
     // Do not modify these 2 lines of code.
    for (register int iter = 0; iter < nIters; iter++) {
        StartTimer();
        /*******************************************************************/
        /*
         * You will likely wish to refactor the work being done in `bodyForce`,
         * as well as the work to integrate the positions.
         */
        bodyForce << <BLOCK_NUM * BLOCK_STEP, BLOCK_SIZE >> > (d_p, dt, nBodies); // compute interbody forces
      /*
       * This position integration cannot occur until this round of `bodyForce` has completed.
       * Also, the next round of `bodyForce` cannot begin until the integration is complete.
       */
        if (iter == nIters - 1)
            hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);//���һ��ʱ��Ƭ��д��CPU
        /*******************************************************************/
        // Do not modify the code in this section.
        const double tElapsed = GetTimer() / 1000.0;
        totalTime += tElapsed;
    }
    double avgTime = totalTime / (double)(nIters);
    float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;
#ifdef ASSESS
    checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
    checkAccuracy(buf, nBodies);
    printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
    salt += 1;
#endif
    /*******************************************************************/
    /*
     * Feel free to modify code below.
     */
    hipFree(buf);
}